#include <stdio.h>
#include <hip/hip_runtime.h>
int devCount;
int myid;
int ihavecuda;
int deviceselector=0;


int main(void) {

    hipGetDeviceCount(&devCount);
    if (devCount == 0) {
        printf("Devcount %4d NONE\n", devCount);
        ihavecuda=0;
    }
    else{
        ihavecuda=1;
        if (devCount >= 1){
            printf("Devcount %4d\n", devCount);
            for (int i = 0; i < devCount; ++i)
            {
                hipDeviceProp_t devProp;
                hipGetDeviceProperties(&devProp, i);
                printf(" devprop name %s i=(%d) \n ", devProp.name, i);
            }
        }
    }


}
